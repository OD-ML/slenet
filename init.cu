#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "headers/slenet_params.h"
#include "headers/load_mnist.h"
#include "headers/Layer.h"
#include "headers/Slenet_shv2.h"

// Layer declarations
Layer *convNet;
Layer *ss1Net;
Layer *fcNet;

float convWeights[CONV_FTRS][CONV_WSIZE][CONV_WSIZE];
float convBias[CONV_FTRS];
float ssWeights[SS_FTRS][SS_WSIZE][SS_WSIZE];
float ssBias[SS_FTRS];
float fcWeights[FC_FTRS][FC_WSIZE];
float fcBias[FC_FTRS];

int count = 0;

float forward_pass(double data[INSIZE][INSIZE]) {
	float *gInput;
	float arr[INSIZE][INSIZE];
	for (int i=0; i<INSIZE; i++)
		for (int j=0; j<INSIZE; j++)
			arr[i][j] = data[i][j];

	// Copying input to Cuda memory
	gpuErrchk(hipMalloc(&gInput, INSIZE*INSIZE*sizeof(float)));
	gpuErrchk(hipMemcpy(gInput, arr, INSIZE*INSIZE*sizeof(float), hipMemcpyDefault));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	// Convolution
	kernel_conv_filter<<<cf_numBlocks, cf_threadPerBlock>>>(
      (float(*)[INSIZE])gInput, 
      (float(*)[CONV_OUTSIZE][CONV_OUTSIZE])convNet->pre_output, 
      (float(*)[CONV_WSIZE][CONV_WSIZE])convNet->weight);
	kernel_conv_bias<<<cb_numBlocks, cb_threadPerBlock>>>(
      (float(*)[CONV_OUTSIZE][CONV_OUTSIZE])convNet->pre_output, 
      convNet->bias);
	kernel_conv_sigmoid<<<cs_numBlocks, cs_threadPerBlock>>>(
      (float(*)[CONV_OUTSIZE][CONV_OUTSIZE])convNet->pre_output, 
      (float(*)[CONV_OUTSIZE][CONV_OUTSIZE])convNet->output);

	// Subsampling
	kernel_ss1_filter<<<ssf_numBlocks, ssf_threadPerBlock>>>(
      (float(*)[CONV_OUTSIZE][CONV_OUTSIZE])convNet->output, 
      (float(*)[SS_OUTSIZE][SS_OUTSIZE])ss1Net->pre_output, 
      (float(*)[SS_WSIZE][SS_WSIZE])ss1Net->weight);
	kernel_ss1_bias<<<ssb_numBlocks, ssb_threadPerBlock>>>(
      (float(*)[SS_OUTSIZE][SS_OUTSIZE])ss1Net->pre_output, 
      ss1Net->bias);
	kernel_ss1_sigmoid<<<sss_numBlocks, sss_threadPerBlock>>>(
      (float(*)[SS_OUTSIZE][SS_OUTSIZE])ss1Net->pre_output, 
      (float(*)[SS_OUTSIZE][SS_OUTSIZE])ss1Net->output);

	// Fully Connected
	kernel_fc1_filter<<<fcfNumBlocks, fcfNthreadPerBlock>>>(
      (float(*)[SS_OUTSIZE][SS_OUTSIZE])ss1Net->output, 
      fcNet->pre_output, 
      (float(*)[FC_WSIZE])fcNet->weight);
	kernel_fc1_bias<<<fcbsNumBlocks, fcbsNthreadPerBlock>>>(fcNet->pre_output, fcNet->bias);
	kernel_fc1_sigmoid<<<fcbsNumBlocks, fcbsNthreadPerBlock>>>(fcNet->pre_output, fcNet->output);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float eltime;
	hipEventElapsedTime(&eltime, start, stop);
	return eltime;
}

int main() {
	int ret; int i;
	mnist_data *dataset = new mnist_data[10000];
	static unsigned int test_cnt;
	
	// load data
	if (ret = mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte", &dataset, &test_cnt) != 0)
		printf("An error occurred: %d \n", ret);
	else
		printf("test_cnt = %d \n", test_cnt); // test_cnt must have the number of test images (i.e., 10K)

	for (int i=0; i < CONV_FTRS; i++)
		for (int j = 0; j < CONV_WSIZE; j++)
			for (int k = 0; k < CONV_WSIZE; k++)
				convWeights[i][j][k] = c1_weight[i][j*CONV_WSIZE+k];
  
	for (int i=0; i < CONV_FTRS; i++)
		convBias[i] = c1_bias[i];

	for (int i=0; i < SS_FTRS; i++)
		for (int j = 0; j < SS_WSIZE; j++)
			for (int k = 0; k < SS_WSIZE; k++)
				ssWeights[i][j][k] = s2_weight[i][j*SS_WSIZE+k];
  
	for (int i=0; i < SS_FTRS; i++)
		ssBias[i] = s2_bias[i];
  
	for (int i=0; i < FC_FTRS; i++)
		for (int j=0; j < FC_WSIZE; j++)
			fcWeights[i][j] = f3_weight[i][j];
  
	for (int i=0; i < FC_FTRS; i++)
		fcBias[i] = f3_bias[i];

	convNet = new Layer(CONV_WSIZE*CONV_WSIZE, CONV_FTRS, CONV_FTRS*CONV_OUTSIZE*CONV_OUTSIZE);
	ss1Net = new Layer(SS_WSIZE*SS_WSIZE, SS_FTRS, CONV_FTRS*SS_OUTSIZE*SS_OUTSIZE);
	fcNet = new Layer(FC_WSIZE, FC_FTRS, FC_OUTSIZE);
  	gpuErrchk(hipMemcpy(convNet->weight, 
                      convWeights, 
                      CONV_WSIZE * CONV_WSIZE * CONV_FTRS * sizeof(float), 
                      hipMemcpyDefault));
	gpuErrchk(hipMemcpy(convNet->bias, 
                      convBias, 
                      CONV_FTRS * sizeof(float), 
                      hipMemcpyDefault));
  	gpuErrchk(hipMemcpy(ss1Net->weight, 
                      ssWeights, 
                      SS_FTRS * SS_WSIZE * SS_WSIZE * sizeof(float), 
                      hipMemcpyDefault));
	gpuErrchk(hipMemcpy(ss1Net->bias, 
                      ssBias, 
                      SS_FTRS * sizeof(float), 
                      hipMemcpyDefault));
	gpuErrchk(hipMemcpy(fcNet->weight, 
                      fcWeights, FC_FTRS * FC_WSIZE * sizeof(float), 
                      hipMemcpyDefault));
	gpuErrchk(hipMemcpy(fcNet->bias, 
                      fcBias, FC_FTRS * sizeof(float), 
                      hipMemcpyDefault));
  
	float time_taken = 0;
	unsigned int error = 0;
	unsigned int max = 0;
	float res[10];
  
	for (i=0; i<10000; i++){
    time_taken += forward_pass(dataset[i].data);
    hipMemcpy(res, fcNet->output, sizeof(float)*10, hipMemcpyDefault);
    for(int j=0; j<10; j++){
      if (res[max] < res[j])
        max = j;
      }
    if (max != dataset[i].label) ++error; // error must have the number of incorrect predictions.
	}
	printf("Error Rate = %f%% (%d out of 10,000)\n", double(error)/double(test_cnt)*100.0, error);
	printf("Accuracy = %.3f%% (%d out of 10,000)\n",
		 100.0 - double(error)/double(test_cnt)*100.0, test_cnt - error);
	printf("Ex time = %f (ms) \n", time_taken);
  
	delete[] dataset;
	delete convNet;
	delete ss1Net;
	delete fcNet;
	return 0;
}
